#include "kernels.cuh"
#include "driver.cuh"
#include <cstdio>
#include <cmath>
#include <cstdlib>
#include <iomanip>

struct matrix {
    int m;
    int n;
    int type;
    float* data;
    bool is_square;
    bool is_symmetric;
};

struct vector {
    int size;
    float* data;
};

float get_sec() {
    struct timeval time;
    gettimeofday(&time, NULL);
    return (1e6 * time.tv_sec + time.tv_usec);
}

float cpu_elapsed_time(float &beg, float &end) { return 1.0e-6 ** (end - beg); }

void cudaCheck(hipError_t error, const char* file, int line) {
    if (error != hipSuccess) {
        printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line, hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
};

void CudaDeviceInfo() {

    int deviceId;

    hipGetDevice(&deviceId);

    hipDeviceProp_t props{};
    hipGetDeviceProperties(&props, deviceId);

    printf("Device ID: %d\n\
    Name: %s\n\
    Compute Capability: %d.%d\n\
    memoryBusWidth: %d\n\
    maxThreadsPerBlock: %d\n\
    maxThreadsPerMultiProcessor: %d\n\
    maxRegsPerBlock: %d\n\
    maxRegsPerMultiProcessor: %d\n\
    totalGlobalMem: %zuMB\n\
    sharedMemPerBlock: %zuKB\n\
    sharedMemPerMultiprocessor: %zuKB\n\
    totalConstMem: %zuKB\n\
    multiProcessorCount: %d\n\
    Warp Size: %d\n",
         deviceId, props.name, props.major, props.minor, props.memoryBusWidth,
         props.maxThreadsPerBlock, props.maxThreadsPerMultiProcessor,
         props.regsPerBlock, props.regsPerMultiprocessor,
         props.totalGlobalMem / 1024 / 1024, props.sharedMemPerBlock / 1024,
         props.sharedMemPerMultiprocessor / 1024, props.totalConstMem / 1024,
         props.multiProcessorCount, props.warpSize);
};

void randomize_matrix(float *mat, int N) {

    struct timeval time {};
    gettimeofday(&time, nullptr);
    srand(time.tv_usec);
    for (int i = 0; i < N; i++) {
        float tmp = (float)(rand() % 5) + 0.01 * (rand() % 5);
        tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
        mat[i] = tmp;
    }
}

void range_init_matrix(float *mat, int N) {
    for (int i = 0; i < N; i++) {
        mat[i] = i;
    }
}

void zero_init_matrix(float *mat, int N) {
    for (int i = 0; i < N; i++) {
        mat[i] = 0.0;
    }
}


void copy_matrix(const float *src, float *dest, int N) {
    int i;
    for (i = 0; src + i && dest + i && i < N; i++) {
        *(dest + i) = *(src + i);
    if (i != N)
        printf("copy failed at %d while there are %d elements in total.\n", i, N);
    }
}

void print_matrix(const float *A, int M, int N, std::ofstream &fs) {
    int i;
    fs << std:: setprecision(2) << std::fixed;
    fs << "[";
    for (i = 0; i < M * N; i++) {
        if ((i + 1) % N == 0)
            fs << std::setw(5) << A[i];
        else 
            fs << std::setw(5) << A[i] << ", ";
        if ((i + 1) % N == 0) {
            if (i + 1 < M * N)
            fs << ";\n";
        }
        fs << "]\n";
    }
}

// compute matrix dims
// return array of [m, n, (1 for square, 2 for non-square)]
int* compute_matrix_dims(int m, int n) {
    int* dims = new int[3];
    dims[0] = m;
    dims[1] = n;
    dims[2] = (m == n) ? 1 : 2;
    return dims;
}


// generate the appropriate matrix on device, using cuRAND when appropriate
void generateMatrix(float* matrix, m , n, is_square, is_symmetric) {

    // allocate memory for the matrix on the device
    float *d_matrix;
    hipMalloc((void**)&d_matrix, m * n * sizeof(float));
    // create the cuRAND generator
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);

    if (is_square) {
        if (is_symmetric) {
            // generate symmetric square matrix using cuRAND
            hiprandGenerateUniform(gen, d_matrix, m * n);
            hiprandDestroyGenerator(gen);
        } else {
            // generate nonsymmetric square matrix using cuRAND
            // TODO: this isn't done yet
            hiprandGenerateUniform(gen, d_matrix, m * n);
            hiprandDestroyGenerator(gen);
            // copy to from device to host
            float *h_matrix = (float*)malloc(m * n * sizeof(float));
            hipMemcpy(h_matrix, d_matrix, m * n * sizeof(float), hipMemcpyDeviceToHost);
            hipFree(d_matrix);
        }
    } else {
        // generate nonsquare matrix without hiprand on the device
        float *h_matrix = (float*)malloc(m * n * sizeof(float));
        for (int i = 0; i < m * n; i++) {
            h_matrix[i] = (float)rand() / RAND_MAX;
        }
        // copy the matrix from device to host 
        hipMemcpy(d_matrix, h_matrix, m * n * sizeof(float), hipMemcpyHostToDevice);
        hipFree(d_matrix);

    }
}

// calls the matrix-vector multiplication kernel
void run_matrix_vector_mul_kernel(matrix, vector) 
